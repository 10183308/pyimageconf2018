#include "hip/hip_runtime.h"
#include <dlib/cuda/cuda_utils.h>
#include <dlib/cuda/cuda_data_ptr.h>
#include "cuda_stuff.h"

// ----------------------------------------------------------------------------------------

// __global__ is a CUDA keyword that means "this function runs on the GPU".  People call
// such functions "kernels".
__global__ void kernel_add_value_to_each_element_simple(float* data, const float value, size_t n)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;    
    if (i < n)
        data[i] += value;
}

void cuda_add_value_to_each_element_simple (
    cuda_data_ptr<float>& img,
    const double value
)
{
    kernel_add_value_to_each_element_simple<<<4096,256>>>(img, value, img.size());
}

// ----------------------------------------------------------------------------------------

__global__ void kernel_add_value_to_each_element_ugly(float* data, const float value, size_t n)
{
    const auto num_cuda_threads = blockDim.x * gridDim.x;
    const auto thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

    // this is called a "grid stride loop" and allows our kernel to efficiently handle
    // arrays of any length, regardless of how many cuda threads we have running.
    for (auto i = thread_idx; i < n; i += num_cuda_threads)
    {
        data[i] += value;
    }
}

void cuda_add_value_to_each_element_ugly (
    cuda_data_ptr<float>& img,
    const double value
)
{
    kernel_add_value_to_each_element_ugly<<<4096,256>>>(img, value, img.size());
}

// ----------------------------------------------------------------------------------------

__global__ void kernel_add_value_to_each_element(float* data, const float value, size_t n)
{
    // This is a nice C++11 way to write the same grid stride loop as above.
    for (auto i : grid_stride_range(0, n))
    {
        data[i] += value;
    }
}

void cuda_add_value_to_each_element (
    cuda_data_ptr<float>& img,
    const double value
)
{
    launch_kernel(kernel_add_value_to_each_element, img, value, img.size());
}

// ----------------------------------------------------------------------------------------

__global__ void kernel_set_to_0(float* out, size_t n)
{
    for (auto i : grid_stride_range(0, n))
        out[i] = 0;
}

__global__ void kernel_dot_product(float* out, const float* a, const float* b, size_t n)
{
    // Parallel sum everything into local temp variables.
    float temp = 0;
    for (auto i : grid_stride_range(0, n))
        temp += a[i]*b[i];
    
    // Each CUDA thread has a temp variable that contains a partial sum.  We need to get
    // them to all add together to out, but we can't just do:
    //   *out += temp
    // because it's illegal for multiple threads to write to the same memory at the same
    // time (as you might expect).  To do this, you need to use special atomic commands
    // that force the appropriate synchronization:
    warp_reduce_atomic_add(*out, temp); // do *out += temp, but legally.
    // There is a good article that goes into "warp reduce atomic" stuff here: https://devblogs.nvidia.com/faster-parallel-reductions-kepler/
}

void cuda_dot_product (
    cuda_data_ptr<float>& out,
    const cuda_data_ptr<float>& a,
    const cuda_data_ptr<float>& b
)
{
    DLIB_CASSERT(a.size() == b.size());
    DLIB_CASSERT(out.size() == 1);

    // tell cuda not to launch more than out.size() threads so we don't waste resources.
    launch_kernel(kernel_set_to_0, max_jobs(out.size()), out, out.size());
    // run the dot product kernel
    launch_kernel(kernel_dot_product, out, a, b, a.size());
}

// ----------------------------------------------------------------------------------------

__global__ void kernel_matrix_vector_multiply (float* out, const float* M, const float* v, size_t nr, size_t nc)
{
    // initialize out to 0
    for (auto r : grid_stride_range_y(0, nr))
        for (auto c : grid_stride_range(0, 1))
            out[r] = 0;

    __syncthreads(); // synchronize threads in block so we don't start the next bit until out is really 0.

    for (auto r : grid_stride_range_y(0, nr))
    {
        float temp = 0;
        for (auto c : grid_stride_range(0, nc))
            temp += M[r*nc+c]*v[c];

        // store the sum into out[r]
        warp_reduce_atomic_add(out[r], temp);
    }
}

// computes out = M*v
void cuda_matrix_vector_multiply (
    cuda_data_ptr<float>& out,
    const cuda_data_ptr<float>& M,
    const cuda_data_ptr<float>& v
)
{
    DLIB_CASSERT(M.size() == out.size()*v.size());

    const auto nr = out.size();
    const auto nc = v.size();
    launch_kernel(kernel_matrix_vector_multiply, max_jobs(nc,nr), out, M, v, nr, nc);
}

// ----------------------------------------------------------------------------------------

/*
    talk about hipSetDevice()

    See also:
    https://devblogs.nvidia.com/even-easier-introduction-cuda/


    I also like the book:
    Professional CUDA C Programming by John Cheng, Max Grossman, and Ty McKercher
*/
